
/* Copyright (c) . All rights reserved. */

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *data1, int *data2, int *X , int *Y , int *Z, int numElements) {

int N = 2, i = 0;

do {
    int t1 = threadIdx.x;
    int t2 = t1 + i; //add.s32 %r2, %r5, %r1;
    int t3 = data1[t2];  //A
    int t4 = 0;  //

    if (t3 != t4) {  //setp.ne.s32	%p1, %r6, 0; split warp
        int t5 = data2[t2]; //B
        if (t5 != t4) {
            X[t2] +=1;  //C
        } else {
            Y[t2] +=2;  //D
        }
        Z[t2] += X[t2]; //E

    } else {
        Z[t2] += 3; //F
    }
    Z[t2] += X[t2];//G
    i++;
    } while (i<N);
}

int main(void) {
  hipError_t err = hipSuccess;

  int numElements = 32;
  size_t size = numElements * sizeof(int);

  int *h_data1_in  = (int *)malloc(size);
  int *h_data2_in  = (int *)malloc(size);
  int *h_data1_out = (int *)malloc(size);
  int *h_data2_out = (int *)malloc(size);
  int *h_data3_out = (int *)malloc(size);

  for (int i = 0; i < numElements; ++i) {
    h_data1_in[i] = i;
    h_data2_in[i] = numElements - i;
  }

  int *d_data1_in = NULL;
  err = hipMalloc((void **)&d_data1_in, size);
  
  int *d_data2_in = NULL;
  err = hipMalloc((void **)&d_data2_in, size);

  int *d_data1_out = NULL;
  err = hipMalloc((void **)&d_data1_out, size);
  int *d_data2_out = NULL;
  err = hipMalloc((void **)&d_data2_out, size);
  int *d_data3_out = NULL;
  err = hipMalloc((void **)&d_data3_out, size);

  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_data1_in,  h_data1_in, size, hipMemcpyHostToDevice);
  err = hipMemcpy(d_data2_in,  h_data2_in, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 32;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_data1_in, d_data2_in, d_data1_out,d_data2_out , d_data3_out,numElements);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  printf("Copy output data from CUDA device to the host memory\n");
  err = hipMemcpy(h_data1_out, d_data1_out, size, hipMemcpyDeviceToHost);
  err = hipMemcpy(h_data2_out, d_data2_out, size, hipMemcpyDeviceToHost);
  err = hipMemcpy(h_data3_out, d_data3_out, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < numElements; i++) {
    printf("%d ", h_data1_out[i]);
    printf("%d ", h_data2_out[i]);
    printf("%d ", h_data3_out[i]);
    }
  printf("\n");

  err = hipFree(d_data1_in);
  err = hipFree(d_data2_in);
  err = hipFree(d_data1_out);
  err = hipFree(d_data2_out);
  err = hipFree(d_data3_out);

  free(h_data1_in);
  free(h_data2_in);
  free(h_data1_out);
  free(h_data2_out);
  free(h_data3_out);

  printf("Done\n");
  return 0;
}

